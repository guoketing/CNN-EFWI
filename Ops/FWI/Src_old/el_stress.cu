#include "hip/hip_runtime.h"
#define d_vx(z,x)  d_vx[(x)*(nz)+(z)]
#define d_vy(z,x)  d_vy[(x)*(nz)+(z)]
#define d_vz(z,x)  d_vz[(x)*(nz)+(z)]
#define d_sxx(z,x) d_sxx[(x)*(nz)+(z)]
#define d_szz(z,x) d_szz[(x)*(nz)+(z)]
#define d_sxz(z,x) d_sxz[(x)*(nz)+(z)]
#define d_sxx_adj(z,x) d_sxx_adj[(x)*(nz)+(z)]
#define d_szz_adj(z,x) d_szz_adj[(x)*(nz)+(z)]
#define d_sxz_adj(z,x) d_sxz_adj[(x)*(nz)+(z)]
#define d_mem_dvz_dz(z,x) d_mem_dvz_dz[(x)*(nz)+(z)]
#define d_mem_dvz_dx(z,x) d_mem_dvz_dx[(x)*(nz)+(z)]
#define d_mem_dvx_dz(z,x) d_mem_dvx_dz[(x)*(nz)+(z)]
#define d_mem_dvx_dx(z,x) d_mem_dvx_dx[(x)*(nz)+(z)]
#define d_Lambda(z,x)     d_Lambda[(x)*(nz)+(z)]
#define d_Mu(z,x)         d_Mu[(x)*(nz)+(z)]
#define d_ave_Mu(z,x)     d_ave_Mu[(x)*(nz)+(z)]
#define d_Den(z,x)        d_Den[(x)*(nz)+(z)]
#define d_LambdaGrad(z,x)     d_LambdaGrad[(x)*(nz)+(z)]
#define d_MuGrad(z,x)         d_MuGrad[(x)*(nz)+(z)]
#include "utilities.h"
#define sh_vz(z,x)  			sh_vz[(x)*(localz)+(z)]
#define sh_vx(z,x)  			sh_vx[(x)*(localz)+(z)]

__global__ void el_stress(float *d_vz, float *d_vx, float *d_szz, \
	float *d_sxx, float *d_sxz, float *d_mem_dvz_dz, float *d_mem_dvz_dx, \
	float *d_mem_dvx_dz, float *d_mem_dvx_dx, float *d_Lambda, float *d_Mu, float *d_ave_Mu,\
	float *d_Den, float *d_K_z, float *d_a_z, float *d_b_z, float *d_K_z_half, \
	float *d_a_z_half, float *d_b_z_half, float *d_K_x, float *d_a_x, float *d_b_x, \
	float *d_K_x_half, float *d_a_x_half, float *d_b_x_half, \
	int nz, int nx, float dt, float dz, float dx, int nPml, int nPad, bool isFor, \
	float *d_szz_adj, float *d_sxx_adj, float *d_sxz_adj, float *d_LambdaGrad, float *d_MuGrad){


  int gidz = blockIdx.x*blockDim.x + threadIdx.x;
  int gidx = blockIdx.y*blockDim.y + threadIdx.y;

  float dvz_dz = 0.0;
  float dvx_dx = 0.0;
  float dvx_dz = 0.0;
  float dvz_dx = 0.0;

  float c1 = 9.0/8.0;
  float c2 = 1.0/24.0;
  // float c1 = coef[0];
  // float c2 = coef[1];

  // float lambda = d_Lambda(gidz,gidx);
  // float mu = d_Mu(gidz,gidx);

  if (isFor) {

		if(gidz>=2 && gidz<=nz-nPad-3 && gidx>=2 && gidx<=nx-3) {

		  dvz_dz = (c1*(d_vz(gidz,gidx)-d_vz(gidz-1,gidx)) - c2*(d_vz(gidz+1,gidx)-d_vz(gidz-2,gidx)))/dz;
		  dvx_dx = (c1*(d_vx(gidz,gidx)-d_vx(gidz,gidx-1)) - c2*(d_vx(gidz,gidx+1)-d_vx(gidz,gidx-2)))/dx;

		  if(gidz<nPml || (gidz>nz-nPml-nPad-1)){
			  d_mem_dvz_dz(gidz,gidx) = d_b_z[gidz]*d_mem_dvz_dz(gidz,gidx) + d_a_z[gidz]*dvz_dz;
			  dvz_dz = dvz_dz / d_K_z[gidz] + d_mem_dvz_dz(gidz,gidx);
			}
			if(gidx<nPml || gidx>nx-nPml-1){
			  d_mem_dvx_dx(gidz,gidx) = d_b_x[gidx]*d_mem_dvx_dx(gidz,gidx) + d_a_x[gidx]*dvx_dx;
			  dvx_dx = dvx_dx / d_K_x[gidx] + d_mem_dvx_dx(gidz,gidx);
			}

			d_szz(gidz,gidx) += ((d_Lambda(gidz,gidx)+2.0*d_Mu(gidz,gidx))*dvz_dz + d_Lambda(gidz,gidx)*dvx_dx) * dt;
		  d_sxx(gidz,gidx) += (d_Lambda(gidz,gidx)*dvz_dz + (d_Lambda(gidz,gidx)+2.0*d_Mu(gidz,gidx))*dvx_dx) * dt;



		  dvx_dz = (c1*(d_vx(gidz+1,gidx)-d_vx(gidz,gidx)) - c2*(d_vx(gidz+2,gidx)-d_vx(gidz-1,gidx)))/dz;
		  dvz_dx = (c1*(d_vz(gidz,gidx+1)-d_vz(gidz,gidx)) - c2*(d_vz(gidz,gidx+2)-d_vz(gidz,gidx-1)))/dx;

		  if(gidz<nPml || (gidz>nz-nPml-nPad-1)){
			  d_mem_dvx_dz(gidz,gidx) = d_b_z_half[gidz]*d_mem_dvx_dz(gidz,gidx) + d_a_z_half[gidz]*dvx_dz;
			  dvx_dz = dvx_dz / d_K_z_half[gidz] + d_mem_dvx_dz(gidz,gidx);
			}
			if(gidx<nPml || gidx>nx-nPml-1){
			  d_mem_dvz_dx(gidz,gidx) = d_b_x_half[gidx]*d_mem_dvz_dx(gidz,gidx) + d_a_x_half[gidx]*dvz_dx;
			  dvz_dx = dvz_dx / d_K_x_half[gidx] + d_mem_dvz_dx(gidz,gidx);
			}

		  d_sxz(gidz,gidx) = d_sxz(gidz,gidx) + d_ave_Mu(gidz,gidx) * (dvx_dz + dvz_dx) * dt;
		}
		else{
			return;
		}
	}

	else {

		// ========================================BACKWARD PROPAGATION====================================
		if(gidz>=nPml && gidz<=nz-nPad-1-nPml && gidx>=nPml && gidx<=nx-1-nPml) {

		  dvz_dz = (c1*(d_vz(gidz,gidx)-d_vz(gidz-1,gidx)) - c2*(d_vz(gidz+1,gidx)-d_vz(gidz-2,gidx)))/dz;
		  dvx_dx = (c1*(d_vx(gidz,gidx)-d_vx(gidz,gidx-1)) - c2*(d_vx(gidz,gidx+1)-d_vx(gidz,gidx-2)))/dx;

		  d_szz(gidz,gidx) -= ((d_Lambda(gidz,gidx)+2.0*d_Mu(gidz,gidx))*dvz_dz + d_Lambda(gidz,gidx)*dvx_dx) * dt;
		  d_sxx(gidz,gidx) -= (d_Lambda(gidz,gidx)*dvz_dz + (d_Lambda(gidz,gidx)+2.0*d_Mu(gidz,gidx))*dvx_dx) * dt;


		  dvx_dz = (c1*(d_vx(gidz+1,gidx)-d_vx(gidz,gidx)) - c2*(d_vx(gidz+2,gidx)-d_vx(gidz-1,gidx)))/dz;
		  dvz_dx = (c1*(d_vz(gidz,gidx+1)-d_vz(gidz,gidx)) - c2*(d_vz(gidz,gidx+2)-d_vz(gidz,gidx-1)))/dx;

			d_sxz(gidz,gidx) -= d_ave_Mu(gidz,gidx) * (dvx_dz + dvz_dx) * dt;
			
			// computate the kernels of lame parameters
			// d_LambdaGrad(gidz,gidx) += (-d_szz_adj(gidz,gidx)*dvz_dz*dt - d_szz_adj(gidz,gidx)*dvx_dx*dt - d_sxx_adj(gidz,gidx)*dvz_dz*dt - d_sxx_adj(gidz,gidx)*dvx_dx*dt) * MEGA;
			d_LambdaGrad(gidz,gidx) += -(d_szz_adj(gidz,gidx)+d_sxx_adj(gidz,gidx)) * (dvz_dz+dvx_dx) * dt;

			d_MuGrad(gidz,gidx) += (-2.0*d_szz_adj(gidz,gidx)*dvz_dz*dt - 2.0*d_sxx_adj(gidz,gidx)*dvx_dx*dt);
			// spray (atomic add)
			if (d_ave_Mu(gidz,gidx) != 0.0) {
				float scale = -d_sxz_adj(gidz,gidx) * (dvx_dz + dvz_dx) * dt \
					* d_ave_Mu(gidz,gidx)/(1.0/d_Mu(gidz,gidx) + 1.0/d_Mu(gidz+1,gidx) \
					+ 1.0/d_Mu(gidz,gidx+1) + 1.0/d_Mu(gidz+1,gidx+1));
				atomicAdd(&d_MuGrad[gidz+nz*gidx], 1.0/pow(d_Mu(gidz,gidx), 2)*scale);
				if (gidz+1<=nz-nPad-1-nPml) 
					atomicAdd(&d_MuGrad[gidz+1+nz*gidx], 1.0/pow(d_Mu(gidz+1,gidx), 2)*scale);
				if (gidx+1<=gidx<=nx-1-nPml) 
					atomicAdd(&d_MuGrad[gidz+nz*(gidx+1)], 1.0/pow(d_Mu(gidz,gidx+1), 2)*scale);
				if (gidz+1<=nz-nPad-1-nPml && gidx+1<=nx-1-nPml)
					atomicAdd(&d_MuGrad[gidz+1+nz*(gidx+1)], 1.0/pow(d_Mu(gidz+1,gidx+1), 2)*scale);
			}

		}
		else{
			return;
		}

	}

}
