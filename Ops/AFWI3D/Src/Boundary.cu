#include "Boundary.h"
#include "Parameter.h"
#include "utilities.h"

Bnd::Bnd(const Parameter &para) {
  withAdj_ = para.withAdj();
  if (withAdj_) {
    nz_ = para.nz();
    nx_ = para.nx();
    ny_ = para.ny();
    nPml_ = para.nPoints_pml();
    nSteps_ = para.nSteps();

    nzBnd_ = nz_ - 2 * nPml_ + 4;
    nxBnd_ = nx_ - 2 * nPml_ + 4;
    nyBnd_ = ny_ - 2 * nPml_ + 4;
    nLayerStore_ = 5;

    len_Bnd_vec_ =
        2 * (nLayerStore_ * (nzBnd_ * nxBnd_) + nLayerStore_ * (nxBnd_ * nyBnd_) + nLayerStore_ * (nyBnd_ * nzBnd_));  // store n layers

    // allocate the boundary vector in the device
    CHECK(hipMalloc((void **)&d_Bnd_u,
                     len_Bnd_vec_ * nSteps_ * sizeof(float)));
  }
}

Bnd::~Bnd() {
  if (withAdj_) {
    CHECK(hipFree(d_Bnd_u));
  }
}

void Bnd::field_from_bnd(float *d_u, int indT) {
  from_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_u, d_Bnd_u, nz_, nx_, ny_, nzBnd_,
                                             nxBnd_, nyBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, nPml_, nSteps_);
}

void Bnd::field_to_bnd(float *d_u, int indT) {
    to_bnd<<<(len_Bnd_vec_ + 31) / 32, 32>>>(d_u, d_Bnd_u, nz_, nx_, ny_, nzBnd_,
                                             nxBnd_, nyBnd_, len_Bnd_vec_, nLayerStore_,
                                             indT, nPml_,  nSteps_);
}