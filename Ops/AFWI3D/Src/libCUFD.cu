// Dongzhuo Li 05/06/2018
#include <chrono>
#include <string>
#include "Boundary.h"
#include "Cpml.h"
#include "Model.h"
#include "Parameter.h"
#include "Src_Rec.h"
#include "utilities.h"
using std::string;

#define VERBOSE
// #define DEBUG
/*
        double misfit
        double *grad_Lambda : gradients of Lambda (lame parameter)
        double *grad_Mu : gradients of Mu (shear modulus)
        double *grad_Den : gradients of density
        double *grad_stf : gradients of source time function
        double *Lambda : lame parameter (Mega Pascal)
        double *Mu : shear modulus (Mega Pascal)
        double *Den : density
        double *stf : source time function of all shots
        int calc_id :
                                        calc_id = 0  -- compute residual
                                        calc_id = 1  -- compute gradient
                                        calc_id = 2  -- compute observation only
        int gpu_id  :   CUDA_VISIBLE_DEVICES
        int group_size: number of shots in the group
        int *shot_ids :   processing shot shot_ids
        string para_fname :  parameter path
        // string survey_fname :  survey file (src/rec) path
        // string data_dir : data directory
        // string scratch_dir : temporary files
*/
extern "C" void cufd(float *misfit, float *grad_Vp, float *grad_stf, const float *Vp,
          const float *stf, int calc_id,
          const int gpu_id, const int group_size, const int *shot_ids,
          const string para_fname) {
  // int deviceCount = 0;
  // CHECK(hipGetDeviceCount (&deviceCount));
  // printf("number of devices = %d\n", deviceCount);
  CHECK(hipSetDevice(gpu_id));
  auto start0 = std::chrono::high_resolution_clock::now();

#ifdef VERBOSE
  printf("Calc_ID = %d\n", calc_id);
#endif
  if (calc_id < 0 || calc_id > 2) {
    printf("Invalid calc_id %d\n", calc_id);
    exit(0);
  }

  // NOTE Read parameter file
  Parameter para(para_fname, calc_id);
  int nz = para.nz();
  int nx = para.nx();
  int ny = para.ny();
  int nShape = nz * nx * ny;
  int nPml = para.nPoints_pml();
  int nPad = para.nPad();
  float dz = para.dz();
  float dx = para.dx();
  float dy = para.dy();
  float dt = para.dt();
  float f0 = para.f0();


  int iSnap = 500;  // 400
  int nrec = 1;
  float win_ratio = 0.005;
  int nSteps = para.nSteps();
  float amp_ratio = 1.0;

  // transpose models and convert to float
  float *fVp;
  fVp = (float *)malloc(nShape * sizeof(float));
  for (int i = 0; i < nz; i++) {
    for (int j = 0; j < nx; j++) {
      for (int k = 0; k < ny; k++) {
        fVp[k * (nx*nz) + j*nz + i] = Vp[i * (nx*ny) + j * (ny) + k];
      }
    }
  }
  Model model(para, fVp);
  // Model model;
  Cpml cpml(para, model);
  Bnd boundaries(para);
  auto startSrc = std::chrono::high_resolution_clock::now();

  Src_Rec src_rec(para, para.survey_fname(), stf, group_size, shot_ids);
  // TODO: group_size -> shot group size
  auto finishSrc = std::chrono::high_resolution_clock::now();
#ifdef VERBOSE
  std::chrono::duration<double> elapsedSrc = finishSrc - startSrc;
  std::cout << "Src_Rec time: " << elapsedSrc.count() << " second(s)"
            << std::endl;
  std::cout << "number of shots " << src_rec.d_vec_z_rec.size() << std::endl;
#endif

  dim3 threads(TX, TY, TZ);
  dim3 blocks((nz + TX - 1) / TX, (nx + TY - 1) / TY, (ny + TZ - 1) / TZ);

  float *d_un, *d_up, *d_uc, *d_un_adj, *d_up_adj, *d_uc_adj;
  float *d_phiz, *d_phix, *d_phiy, *d_psci;
  float *d_l2Obj_temp;
  float *h_l2Obj_temp = nullptr;
  h_l2Obj_temp = (float *)malloc(sizeof(float));
  float h_l2Obj = 0.0;
  float *d_gauss_amp;
  float *d_data;
  float *d_data_obs;
  float *d_res;

  float *d_obs_normfact, *d_cal_normfact, *d_cross_normfact;
  CHECK(hipMalloc((void **)&d_un, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_up, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_uc, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_phiz, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_phix, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_phiy, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_psci, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_un_adj, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_up_adj, nShape * sizeof(float)));
  CHECK(hipMalloc((void **)&d_uc_adj, nShape * sizeof(float)));

  CHECK(hipMalloc((void **)&d_l2Obj_temp, 1 * sizeof(float)));
  CHECK(hipMalloc((void **)&d_gauss_amp, 729 * sizeof(float)));
  dim3 blocks3(1, 1, (9 + TZ - 1) / TZ);
  src_rec_gauss_amp<<<blocks3, threads>>>(d_gauss_amp, 9, 9, 9);

  float *h_snap, *h_snap_back, *h_snap_adj;
  h_snap = (float *)malloc(nShape * sizeof(float));
  h_snap_back = (float *)malloc(nShape * sizeof(float));
  h_snap_adj = (float *)malloc(nShape * sizeof(float));

  hipStream_t *streams = (hipStream_t *)malloc(group_size * sizeof(hipStream_t));
    printf("a");
  auto finish0 = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed0 = finish0 - start0;
#ifdef VERBOSE
  std::cout << "Initialization time: " << elapsed0.count() << " second(s)"
            << std::endl;
#endif

  auto start = std::chrono::high_resolution_clock::now();

  // NOTE Processing Shot
  for (int iShot = 0; iShot < group_size; iShot++) {
      #ifdef VERBOSE
        printf("	Processing shot %d\n", shot_ids[iShot]);
      #endif
    CHECK(hipStreamCreate(&streams[iShot]));
  
    intialArrayGPU<<<blocks, threads>>>(d_up, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_uc, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_un, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_phiz, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_phix, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_phiy, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_psci, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_up_adj, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_uc_adj, nz, nx, ny, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_un_adj, nz, nx, ny, 0.0);

    nrec = src_rec.vec_nrec.at(iShot);

    dim3 threads2(32, 32);
    dim3 blocks2((nSteps + 32 - 1) / 32, (nrec + 32 - 1) / 32);

    CHECK(hipMalloc((void **)&d_data, nrec * nSteps * sizeof(float)));
    intial2dArrayGPU<<<blocks2, threads2>>>(d_data, nSteps, nrec, 0.0);
    if (para.if_res()) {
      fileBinLoad(src_rec.vec_data_obs.at(iShot), nSteps * nrec,
                  para.data_dir_name() + "/Shot" +
                      std::to_string(shot_ids[iShot]) + ".bin");
      CHECK(hipMalloc((void **)&d_data_obs, nrec * nSteps * sizeof(float)));
      CHECK(hipMalloc((void **)&d_res, nrec * nSteps * sizeof(float)));
      intial2dArrayGPU<<<blocks2, threads2>>>(d_data_obs, nSteps, nrec, 0.0);
      intial2dArrayGPU<<<blocks2, threads2>>>(d_res, nSteps, nrec, 0.0);
      CHECK(hipMemcpyAsync(d_data_obs, src_rec.vec_data_obs.at(iShot),
                            nrec * nSteps * sizeof(float),
                            hipMemcpyHostToDevice, streams[iShot]));
            // initialize normalization factors
      if (para.if_cross_misfit()) {
        CHECK(hipMalloc((void **)&d_obs_normfact, nrec * sizeof(float)));
        CHECK(hipMalloc((void **)&d_cal_normfact, nrec * sizeof(float)));
        CHECK(hipMalloc((void **)&d_cross_normfact, nrec * sizeof(float)));
        intial2dArrayGPU<<<1, 512>>>(d_obs_normfact, nrec, 1, 0.0);
        intial2dArrayGPU<<<1, 512>>>(d_cal_normfact, nrec, 1, 0.0);
        intial2dArrayGPU<<<1, 512>>>(d_cross_normfact, nrec, 1, 0.0);
      }
    }
    // ------------------------ time loop ----------------------------
    for (int it = 0; it < nSteps - 1; it++) {
      // =========================== acoustic ==================================

      // get snapshot at time it
        if (it == 700 && iShot == 0) {
         CHECK(hipMemcpy(h_snap, d_up, nShape * sizeof(float), hipMemcpyDeviceToHost));
         fileBinWrite(h_snap, nShape, "SnapGPU_for_.bin");
        printf("saved the forward wavefield!");
      }

      if (para.withAdj()) {
         // save and record from the beginning
         boundaries.field_from_bnd(d_up, it);
      }

      propagate<<<blocks, threads>>>(
                d_un, d_uc, d_up, d_phiz, d_phix, d_phiy, d_psci,
                model.d_Vp, cpml.d_sigma_z, cpml.d_sigma_x,
                cpml.d_sigma_y, nz, nx, ny, dt, dz, dx, dy, nPml, true);

      add_source<<<blocks3, threads>>>(d_un, src_rec.vec_source.at(iShot)[it],
                                 nz, nx, true, src_rec.vec_z_src.at(iShot),
                                 src_rec.vec_x_src.at(iShot), src_rec.vec_y_src.at(iShot), dt, d_gauss_amp);

      exchange_wavefield<<<blocks, threads>>>(d_un, d_uc, d_up, nz, nx);

      recording<<<(nrec + 31) / 32, 32>>>(
            d_uc, nz, nx, d_data, iShot, it + 1, nSteps, nrec,
            src_rec.d_vec_z_rec.at(iShot), src_rec.d_vec_x_rec.at(iShot), src_rec.d_vec_y_rec.at(iShot));
    }  // end of forward time loop

    if (!para.if_res()) {
      CHECK(hipMemcpyAsync(src_rec.vec_data.at(iShot), d_data,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // test
    }

      #ifdef DEBUG
    fileBinWrite(h_snap, nz * nx * ny, "SnapGPU.bin");
      #endif

    // compute residuals
    if (para.if_res()) {
      dim3 blocksT((nSteps + TX - 1) / TX, (nrec + TY - 1) / TY);

      // windowing
      if (para.if_win()) {
        cuda_window<<<blocksT, threads>>>(
            nSteps, nrec, dt, src_rec.d_vec_win_start.at(iShot),
            src_rec.d_vec_win_end.at(iShot), src_rec.d_vec_weights.at(iShot),
            src_rec.vec_srcweights.at(iShot), win_ratio, d_data_obs);
        cuda_window<<<blocksT, threads>>>(
            nSteps, nrec, dt, src_rec.d_vec_win_start.at(iShot),
            src_rec.d_vec_win_end.at(iShot), src_rec.d_vec_weights.at(iShot),
            src_rec.vec_srcweights.at(iShot), win_ratio, d_data);
      } else {
        cuda_window<<<blocksT, threads>>>(nSteps, nrec, dt, win_ratio,
                                          d_data_obs);
        cuda_window<<<blocksT, threads>>>(nSteps, nrec, dt, win_ratio, d_data);
      }

      // filtering
      if (para.if_filter()) {
        bp_filter1d(nSteps, dt, nrec, d_data_obs, para.filter());
        bp_filter1d(nSteps, dt, nrec, d_data, para.filter());
      }

      // normalization 09/26/2019
      if (para.if_cross_misfit()) {
        cuda_find_normfact<<<nrec, 512>>>(nSteps, nrec, d_data_obs, d_data_obs,
                                          d_obs_normfact);
        cuda_find_normfact<<<nrec, 512>>>(nSteps, nrec, d_data, d_data,
                                          d_cal_normfact);
        cuda_find_normfact<<<nrec, 512>>>(nSteps, nrec, d_data_obs, d_data,
                                          d_cross_normfact);
      }

      // Calculate source update and filter calculated data
      if (para.if_src_update()) {
        amp_ratio =
            source_update(nSteps, dt, nrec, d_data_obs, d_data,
                          src_rec.d_vec_source.at(iShot), src_rec.d_coef);
        printf("	Source update => Processing shot %d, amp_ratio = %f\n",
               iShot, amp_ratio);
      }
      amp_ratio = 1.0;  // amplitude not used, so set to 1.0

      // objective function
      if (!para.if_cross_misfit()) {
        gpuMinus<<<blocksT, threads>>>(d_res, d_data_obs, d_data, nSteps, nrec);
        cuda_cal_objective<<<1, 512>>>(d_l2Obj_temp, d_res, nSteps * nrec);
      } else {
        cuda_normal_misfit<<<1, 512>>>(nrec, d_cross_normfact, d_obs_normfact,
                                       d_cal_normfact, d_l2Obj_temp,
                                       src_rec.d_vec_weights.at(iShot),
                                       src_rec.vec_srcweights.at(iShot));
      }

      CHECK(hipMemcpy(h_l2Obj_temp, d_l2Obj_temp, sizeof(float),
                       hipMemcpyDeviceToHost));
      h_l2Obj += h_l2Obj_temp[0];

      //  update source again (adjoint)
      if (para.if_src_update()) {
        source_update_adj(nSteps, dt, nrec, d_res, amp_ratio, src_rec.d_coef);
      }

      // compute negative adjoint source for the normalization objective
      // function 09/26/2019
      if (para.if_cross_misfit()) {
        cuda_normal_adjoint_source<<<blocksT, threads>>>(
            nSteps, nrec, d_obs_normfact, d_cal_normfact, d_cross_normfact,
            d_data_obs, d_data, d_res, src_rec.d_vec_weights.at(iShot),
            src_rec.vec_srcweights.at(iShot));
      }

      // filtering again (adjoint)
      if (para.if_filter()) {
        bp_filter1d(nSteps, dt, nrec, d_res, para.filter());
      }
      // windowing again (adjoint)
      if (para.if_win()) {
        cuda_window<<<blocksT, threads>>>(
            nSteps, nrec, dt, src_rec.d_vec_win_start.at(iShot),
            src_rec.d_vec_win_end.at(iShot), src_rec.d_vec_weights.at(iShot),
            src_rec.vec_srcweights.at(iShot), win_ratio, d_res);
      } else {
        cuda_window<<<blocksT, threads>>>(nSteps, nrec, dt, win_ratio, d_res);
      }

      CHECK(hipMemcpyAsync(src_rec.vec_res.at(iShot), d_res,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // test

      CHECK(hipMemcpyAsync(src_rec.vec_data.at(iShot), d_data,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // test
      CHECK(hipMemcpyAsync(src_rec.vec_data_obs.at(iShot), d_data_obs,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // save preconditioned observed
      CHECK(hipMemcpy(src_rec.vec_source.at(iShot),
                       src_rec.d_vec_source.at(iShot), nSteps * sizeof(float),
                       hipMemcpyDeviceToHost));

    }
    // =================
    hipDeviceSynchronize();


    if (para.withAdj()) {
      // --------------------- Backward ----------------------------
      // initialization
      intialArrayGPU<<<blocks, threads>>>(d_un_adj, nz, nx, ny, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_up_adj, nz, nx, ny, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_uc_adj, nz, nx, ny, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_phiz, nz, nx, ny, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_phix, nz, nx, ny, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_phix, nz, nx, ny, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_psci, nz, nx, ny, 0.0);
      intial2dArrayGPU<<<blocks, threads>>>(model.d_StfGrad, nSteps, 1, 0.0);
      initialArray(model.h_StfGrad, nSteps, 0.0);

      propagate<<<blocks, threads>>>(
          d_un_adj, d_uc_adj, d_up_adj, d_phiz, d_phix, d_phiy, d_psci,
          model.d_Vp, cpml.d_sigma_z, cpml.d_sigma_x, cpml.d_sigma_y, nz, nx, ny, dt, dz, dx, dy, nPml, true);

      res_injection<<<(nrec + 31) / 32, 32>>>(
          d_un_adj, nz, nx, d_res, nSteps - 1, dt, nSteps, nrec,
          src_rec.d_vec_z_rec.at(iShot), src_rec.d_vec_x_rec.at(iShot), src_rec.d_vec_y_rec.at(iShot));

      exchange_wavefield<<<blocks, threads>>>(d_un_adj, d_uc_adj, d_up_adj, nz, nx);

      image_condition<<<blocks, threads>>>(d_uc, d_uc_adj, nz, nx, ny, dz, dx, dy, nPml, model.d_Vp, model.d_VpGrad);

      for (int it = nSteps - 2; it >= 0; it--) {
        // source time function kernels
        // source_grad<<<1, 1>>>(d_szz_adj, d_sxx_adj, nz, model.d_StfGrad, it, dt,
        //                       src_rec.vec_z_src.at(iShot),
        //                       src_rec.vec_x_src.at(iShot),
        //                       src_rec.vec_src_rxz.at(iShot));

        exchange_wavefield<<<blocks, threads>>>(d_up, d_uc, d_un, nz, nx);

        add_source<<<blocks3, threads>>>(d_un, src_rec.vec_source.at(iShot)[it],
                                 nz, nx, false, src_rec.vec_z_src.at(iShot),
                                 src_rec.vec_x_src.at(iShot), src_rec.vec_y_src.at(iShot), dt, d_gauss_amp);

        propagate<<<blocks, threads>>>(
                d_un, d_uc, d_up, d_phiz, d_phix, d_phiy, d_psci,
                model.d_Vp, cpml.d_sigma_z, cpml.d_sigma_x, cpml.d_sigma_y, nz, nx, ny, dt, dz, dx, dy, nPml, false);

        boundaries.field_to_bnd(d_up, it);


        if (it == 700 && iShot == 0) {
                 CHECK(hipMemcpy(h_snap, d_up, nShape * sizeof(float), hipMemcpyDeviceToHost));
                 fileBinWrite(h_snap, nShape, "SnapGPU_back_.bin");
                 CHECK(hipMemcpy(h_snap_adj, d_up_adj, nShape * sizeof(float), hipMemcpyDeviceToHost));
                 fileBinWrite(h_snap_adj, nShape, "SnapGPU_adj_.bin");
                 printf("saved the backward wavefield!");
                exit(0);
        }


        propagate<<<blocks, threads>>>(
          d_un_adj, d_uc_adj, d_up_adj, d_phiz, d_phix, d_phiy, d_psci,
          model.d_Vp, cpml.d_sigma_z, cpml.d_sigma_x, cpml.d_sigma_y, nz, nx, ny, dt, dz, dx, dy, nPml, true);

        res_injection<<<(nrec + 31) / 32, 32>>>(
          d_un_adj, nz, nx, d_res, it, dt, nSteps, nrec,
          src_rec.d_vec_z_rec.at(iShot), src_rec.d_vec_x_rec.at(iShot), src_rec.d_vec_y_rec.at(iShot));

        exchange_wavefield<<<blocks, threads>>>(d_un_adj, d_uc_adj, d_up_adj, nz, nx);

        image_condition<<<blocks, threads>>>(d_uc, d_uc_adj, nz, nx, ny, dz, dx, dy, nPml, model.d_Vp, model.d_VpGrad);

        // if (it == iSnap && iShot == 0) {
        //   CHECK(hipMemcpy(h_snap_back, d_up, nShape * sizeof(float),
        //                    hipMemcpyDeviceToHost));
        //   CHECK(hipMemcpy(h_snap_adj, d_up_adj, nShape * sizeof(float),
        //                    hipMemcpyDeviceToHost));
        // }
        /*
        if (it % 10 == 0 && iShot == 0) {
          CHECK(hipMemcpy(h_snap_adj, d_up_adj, nShape * sizeof(float),
                            hipMemcpyDeviceToHost));
           fileBinWrite(h_snap_adj, nShape,
                        "SnapGPU_adj_" + std::to_string(it) + ".bin");
           CHECK(hipMemcpy(h_snap, d_up, nShape * sizeof(float),
                            hipMemcpyDeviceToHost));
           fileBinWrite(h_snap, nShape,
                       "SnapGPU_" + std::to_string(it) + ".bin");
        }
        */
      }  // the end of backward time loop
      #ifdef DEBUG
        fileBinWrite(h_snap_back, nShape, "SnapGPU_back.bin");
        fileBinWrite(h_snap_adj, nShape, "SnapGPU_adj.bin");
      #endif
      // transfer source gradient to cpu
      CHECK(hipMemcpy(model.h_StfGrad, model.d_StfGrad, nSteps * sizeof(float),
                       hipMemcpyDeviceToHost));
      for (int it = 0; it < nSteps; it++) {
        grad_stf[iShot * nSteps + it] = model.h_StfGrad[it];
      }
    }  // end bracket of if adj
    CHECK(hipFree(d_data));
    if (para.if_res()) {
      CHECK(hipFree(d_data_obs));
      CHECK(hipFree(d_res));
       if (para.if_cross_misfit()) {
         CHECK(hipFree(d_obs_normfact));
         CHECK(hipFree(d_cal_normfact));
         CHECK(hipFree(d_cross_normfact));
       }
    }

  }  // the end of shot loop
  auto finish = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = finish - start;
#ifdef VERBOSE
  std::cout << "Elapsed time: " << elapsed.count() << " second(s)."
            << std::endl;
#endif

  if (para.withAdj()) {
    // transfer gradients to cpu
    CHECK(hipMemcpy(model.h_VpGrad, model.d_VpGrad,
                     nShape * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < nz; i++) {
      for (int j = 0; j < nx; j++) {
        for (int k = 0; k < ny; k++){
        grad_Vp[i * (nx*ny) + j * (ny) + k] = model.h_VpGrad[k * (nz * nx) + j * nz + i];
          }
      }
    }
#ifdef DEBUG
    fileBinWrite(model.h_VpGrad, nShape, "VpGradient.bin");
#endif
  }

  if (!para.if_res()) {
    startSrc = std::chrono::high_resolution_clock::now();
    for (int iShot = 0; iShot < group_size; iShot++) {

      fileBinWrite(src_rec.vec_data.at(iShot),
                   nSteps * src_rec.vec_nrec.at(iShot),
                   para.data_dir_name() + "/Shot" +
                       std::to_string(shot_ids[iShot]) + ".bin");
    }
    finishSrc = std::chrono::high_resolution_clock::now();
#ifdef VERBOSE
    elapsedSrc = finishSrc - startSrc;
    std::cout << "Obs data saving time: " << elapsedSrc.count() << " second(s)"
              << std::endl;
#endif
  }

  // #ifdef DEBUG
  //   std::cout << "cufd--" << __LINE__ << std::endl;
  // #endif

  // output residual
  if (para.if_res()) {
    h_l2Obj = 0.5 * h_l2Obj;  // DL 02/21/2019 (need to make misfit accurate
                              // here rather than in the script)
#ifdef VERBOSE
    std::cout << "Total l2 residual = " << std::to_string(h_l2Obj) << std::endl;
    std::cout << "calc_id = " << calc_id << std::endl;
#endif
    *misfit = h_l2Obj;
  }

  free(h_l2Obj_temp);

  free(h_snap);

  free(h_snap_back);

  free(h_snap_adj);

  free(fVp);

  // destroy the streams
  for (int iShot = 0; iShot < group_size; iShot++)
    CHECK(hipStreamDestroy(streams[iShot]));

  hipFree(d_un);
  hipFree(d_uc);
  hipFree(d_up);
  hipFree(d_phiz);
  hipFree(d_phix);
  hipFree(d_phiy);
  hipFree(d_psci);
  hipFree(d_up_adj);
  hipFree(d_uc_adj);
  hipFree(d_un_adj);
  hipFree(d_l2Obj_temp);
  hipFree(d_gauss_amp);

#ifdef VERBOSE
  std::cout << "Done!" << std::endl;
#endif
}
